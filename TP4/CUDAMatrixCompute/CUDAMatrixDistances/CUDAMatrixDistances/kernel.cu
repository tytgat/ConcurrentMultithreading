#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#define N 5
#define M 3

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
__global__ void MatAdd(int A[][M], int C[][N]) {
	int i = threadIdx.x;
	int j = threadIdx.y;
	int k = blockIdx.x; 
	int val = (A[k][i] - A[k][j]) * (A[k][i] - A[k][j]);
	extern __shared__ int sharedC[N][N];
	sharedC[i][j] = C[i][j];

	if (i == 0 && j == 1)
		printf("[%d - %d] - %d => %d + %d -- (%d-%d)^2\n", i, j, k, val, sharedC[i][j], A[k][i], A[k][j]);

	__syncthreads();
	sharedC[i][j] += val;
	__syncthreads();
	
	C[i][j] = sharedC[i][j];
}

int main() {

	int A[N][M] = { {0,1,1},{4,0,2},{3,1,1},{0,0,0},{2,1,2} };
	int C[N][N];
	int i, j;
	/*for (i = 0; i < N; i++) {
		for (j = 0; j < M; j++) {
			A[i][j] = i + j;
			printf("%d ", A[i][j]);
		}
		printf("\n");
	}*/

	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			C[i][j] = 0;
		}
	}

	int(*pA)[M], (*pC)[N];

	hipMalloc((void**)&pA, (N*M) * sizeof(int));
	hipMalloc((void**)&pC, (N*N) * sizeof(int));

	hipMemcpy(pA, A, (N*M) * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(pC, C, (N*N) * sizeof(int), hipMemcpyHostToDevice);

	int numBlocks = N;
	dim3 threadsPerBlock(N,N);
	int k;
	//for (k = 0; k < N; k++) {
		MatAdd <<<numBlocks, threadsPerBlock >> > (pA, pC);
	//}

	hipMemcpy(C, pC, (N*N) * sizeof(int), hipMemcpyDeviceToHost);

	printf("C = \n");
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			printf("%d ", C[i][j]);
		}
		printf("\n");
	}

	hipFree(pA);
	hipFree(pC);

	printf("\n");

	return 0;
}