#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <time.h>

//#define BLOCK_SIZE 4

#define N 256
#define BLOCK_SIZE 3

__global__ void matMult(float * a, float * d, int n){
	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x; // y index
	int ty = threadIdx.y;

	int ia = n * BLOCK_SIZE * by + n * ty; // index A 
	int ib = BLOCK_SIZE * bx + tx; //index B
	int ic = ia + ib;

	int indexLst = ic;
	int i = indexLst / n;
	int j = indexLst % n;

	float sum = 0.0f;
	__shared__ float ai[BLOCK_SIZE][BLOCK_SIZE+1];
	__shared__ float aj[BLOCK_SIZE][BLOCK_SIZE+1];

	for (int k = 0; k < BLOCK_SIZE; k++) {
		ai[i][k] = a[k + i * BLOCK_SIZE];
		aj[j][k] = a[k + j * BLOCK_SIZE];
	}
	__syncthreads();
	for (int l = 0; l < BLOCK_SIZE; l++) {
		sum += (ai[i][l] - aj[j][l]) * (ai[i][l] - aj[j][l]);
	}
	__syncthreads();
	d[indexLst] = sum;
}

int main() {
	//define memory size
	int numBytes = N * N * sizeof(float); 

	float h_A[N*BLOCK_SIZE];
	float h_D[N*N];

	/*init matrix*/ 
	int i = 0;
	for (i = 0; i < N*BLOCK_SIZE; i++) {
		h_A[i] = (float)i+1;
	}
	for (i = 0; i < N*N; i++) {
		h_D[i] = 0.0;
	}
	

	//assign variable for device
	float * d_A;
	float * d_D;

	// allocate device memory
	hipMalloc((void**)&d_A, numBytes);
	hipMalloc((void**)&d_D, numBytes);

	// set kernel launch configuration
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N/ BLOCK_SIZE,N/ BLOCK_SIZE);
	
	//copy data from host to device
	hipMemcpy(d_A, h_A, numBytes, hipMemcpyHostToDevice);
	
	clock_t begin = clock();
	//kernel launch
	matMult <<<blocks, threads >> > (d_A, d_D, N);
	clock_t end = clock();
	double time_spent = (double)(end - begin);
	printf("%d", time_spent);
	//copy data from device to host
	hipMemcpy(h_D, d_D, numBytes, hipMemcpyDeviceToHost);

	/*for (i = 0; i < N*N; i++) {
		if (i%N == 0) {
			printf("\n");
		}
		printf("%.1f,\t", h_D[i]);
	}*/

	//memory free
	hipFree(d_A);
	hipFree(d_D);


	return 0;

}